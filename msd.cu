#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <iostream>
#include <vector>
#include <math.h>
#include <fstream> 
void checkCUDAError(const char *msg);
#include <hip/hip_runtime.h>
using namespace std;


// --------------------INPUT DATA---------------------
const int Nx = 24, Ny = 120, Nz = 20; // Number of mass points
float maxtime = 60; // End time [sec]
const int Nstep = 1200; // Number of time steps
__device__ __constant__ float dt = 0.05; // maxtime / Nstep; // Time step size [sec]
float dtcpu=0.05;

const int xlength =  (4 + 2 * Nx)*(2 + Ny)*(2 + Nz); // Solution array in x-direction
const int ylength =  (2 + Nx)*(4 + 2 * Ny)*(2 + Nz); // Solution array in y-direction
const int zlength =  (2 + Nx)*(2 + Ny)*(4 + 2 * Nz); // Solution array in z-direction
const int masslength = Nx * Ny * Nz;
const int kxlength = Nz * Ny * (Nx + 1);
const int kylength = Nz * Nx * (Ny + 1);
const int kzlength = Ny * Nx * (Nz + 1);
const int bxlength = kxlength;
const int bylength = kylength;
const int bzlength = kzlength;






//------------------------DEVICE FUNCTIONS----------------------------//
//x-displacement
__device__ float fxx(int n, int i, int j, int k, float*xold)
{
	return xold[ (Ny + 2)*(4 + 2*Nx) + (k - 1)*(Ny + 2)*(4 + 2*Nx) + 4 +
		2*Nx + (i - 1)*(4 + 2*Nx) + 2 + (2*j - 1)-1];
}

//x-velocity
__device__ float fvx(int n, int i, int j, int k, float*xold)
{
	return xold[ (Ny + 2)*(4 + 2*Nx) + (k - 1)*(Ny + 2)*(4 + 2*Nx) + 4 +
		2*Nx + (i - 1)*(4 + 2*Nx) + 2 + (2*j)-1];
}

//y-displacement
__device__ float fyy(int n, int i, int j, int k, float*yold)
{
	return yold[ (Nx + 2)*(4 + 2*Ny) + (k - 1)*(Nx + 2)*(4 + 2*Ny) + 4 +
		2*Ny + (j - 1)*(4 + 2*Ny) + 2 + (2*i - 1)-1];
}

//y-velocity
__device__ float fvy(int n, int i, int j, int k, float*yold)
{
	return yold[ (Nx + 2)*(4 + 2 * Ny) + (k - 1)*(Nx + 2)*(4 + 2 * Ny) + 4 +
		2 * Ny + (j - 1)*(4 + 2 * Ny) + 2 + (2 * i)-1];
}

//z-displacement
__device__ float fzz(int n, int i, int j, int k, float*zold)
{
	return zold[ (Nx + 2)*(4 + 2*Nz) + (i - 1)*(Nx + 2)*(4 + 2*Nz) + 4 +
		2*Nz + (j - 1)*(4 + 2*Nz) + 2 + (2*k - 1)-1];
}

//z-velocity
__device__ float fvz(int n, int i, int j, int k, float*zold)
{
	return zold[ (Nx + 2)*(4 + 2 * Nz) + (i - 1)*(Nx + 2)*(4 + 2 * Nz) + 4 +
		2 * Nz + (j - 1)*(4 + 2 * Nz) + 2 + (2 * k)-1];
}

//mass
__device__ float fm(int i, int j, int k, float*m)
{
	return m[(k - 1)*Ny*Nx + (i - 1)*Nx + j-1];
}

//x-stiffness
__device__ float fkx(int i, int j, int k, float*kx)
{
	return kx[(k - 1)*Ny*(Nx + 1) + (i - 1)*(Nx + 1) + j-1];
}

//y-stiffness
__device__ float fky(int i, int j, int k, float*ky)
{
	return ky[(k - 1)*Nx*(Ny + 1) + (i - 1)*Nx + j-1];
}

//z-stiffness
__device__ float fkz(int i, int j, int k, float*kz)
{
	return kz[(k - 1)*Nx*Ny + (i - 1)*Nx + j-1];
}

//x-damping
__device__ float fbx(int i, int j, int k, float*bx)
{
	return bx[(k - 1)*Ny*(Nx + 1) + (i - 1)*(Nx + 1) + j-1];
}

//y-damping
__device__ float fby(int i, int j, int k, float*by)
{
	return by[(k - 1)*Nx*(Ny + 1) + (i - 1)*Nx + j-1];
}

//z-damping
__device__ float fbz(int i, int j, int k, float*bz)
{
	return bz[(k - 1)*Nx*Ny + (i - 1)*Nx + j-1];
}

//x-force
__device__ float fFx(int i, int j, int k, float*Fx)
{
	return Fx[(k - 1)*Ny*Nx + (i - 1)*Nx + j-1];
}

//y-force
__device__ float fFy(int i, int j, int k, float*Fy)
{
	return Fy[(k - 1)*Ny*Nx + (i - 1)*Nx + j-1];
}

//z-force
__device__ float fFz(int i, int j, int k, float*Fz)
{
	return Fz[(k - 1)*Ny*Nx + (i - 1)*Nx + j-1];
}

//x-acceleration
__device__ float ax(int i, int j, int k, float*Fx, float*xold, float*kx, float*ky, float*kz, float*bx, float*by, float*bz, float*m)
{
	return (fFx(i, j, k, Fx) - fby(i, j, k, by)*(-fvx(1, -1 + i, j, k, xold) + fvx(1, i, j, k, xold)) -
		fbx(i, j, k, bx)*(-fvx(1, i, -1 + j, k, xold) + fvx(1, i, j, k, xold)) - fbz(i, j, k, bz)*(-fvx(1, i, j, -1 + k, xold) + fvx(1, i, j, k, xold)) +
		fbz(i, j, 1 + k, bz)*(-fvx(1, i, j, k, xold) + fvx(1, i, j, 1 + k, xold)) +
		fbx(i, 1 + j, k, bx)*(-fvx(1, i, j, k, xold) + fvx(1, i, 1 + j, k, xold)) +
		fby(1 + i, j, k, by)*(-fvx(1, i, j, k, xold) + fvx(1, 1 + i, j, k, xold)) -
		fky(i, j, k, ky)*(-fxx(1, -1 + i, j, k, xold) + fxx(1, i, j, k, xold)) - fkx(i, j, k, kx)*(-fxx(1, i, -1 + j, k, xold) + fxx(1, i, j, k, xold)) -
		fkz(i, j, k, kz)*(-fxx(1, i, j, -1 + k, xold) + fxx(1, i, j, k, xold)) +
		fkz(i, j, 1 + k, kz)*(-fxx(1, i, j, k, xold) + fxx(1, i, j, 1 + k, xold)) +
		fkx(i, 1 + j, k, kx)*(-fxx(1, i, j, k, xold) + fxx(1, i, 1 + j, k, xold)) +
		fky(1 + i, j, k, ky)*(-fxx(1, i, j, k, xold) + fxx(1, 1 + i, j, k, xold))) / fm(i, j, k, m);
}

//y-acceleration
__device__ float ay(int i, int j, int k, float*Fy, float*yold, float*kx, float*ky, float*kz, float*bx, float*by, float*bz, float*m)
{
	return (fFy(i, j, k, Fy) - fby(i, j, k, by)*(-fvy(1, -1 + i, j, k, yold) + fvy(1, i, j, k, yold)) -
		fbx(i, j, k, bx)*(-fvy(1, i, -1 + j, k, yold) + fvy(1, i, j, k, yold)) - fbz(i, j, k, bz)*(-fvy(1, i, j, -1 + k, yold) + fvy(1, i, j, k, yold)) +
		fbz(i, j, 1 + k, bz)*(-fvy(1, i, j, k, yold) + fvy(1, i, j, 1 + k, yold)) +
		fbx(i, 1 + j, k, bx)*(-fvy(1, i, j, k, yold) + fvy(1, i, 1 + j, k, yold)) +
		fby(1 + i, j, k, by)*(-fvy(1, i, j, k, yold) + fvy(1, 1 + i, j, k, yold)) -
		fky(i, j, k, ky)*(-fyy(1, -1 + i, j, k, yold) + fyy(1, i, j, k, yold)) - fkx(i, j, k, kx)*(-fyy(1, i, -1 + j, k, yold) + fyy(1, i, j, k, yold)) -
		fkz(i, j, k, kz)*(-fyy(1, i, j, -1 + k, yold) + fyy(1, i, j, k, yold)) +
		fkz(i, j, 1 + k, kz)*(-fyy(1, i, j, k, yold) + fyy(1, i, j, 1 + k, yold)) +
		fkx(i, 1 + j, k, kx)*(-fyy(1, i, j, k, yold) + fyy(1, i, 1 + j, k, yold)) +
		fky(1 + i, j, k, ky)*(-fyy(1, i, j, k, yold) + fyy(1, 1 + i, j, k, yold))) / fm(i, j, k, m);
}

//z-acceleration
__device__ float az(int i, int j, int k, float*Fz, float*zold, float*kx, float*ky, float*kz, float*bx, float*by, float*bz, float*m)
{
	return (fFz(i, j, k, Fz) - fby(i, j, k, by)*(-fvz(1, -1 + i, j, k, zold) + fvz(1, i, j, k, zold)) -
		fbx(i, j, k, bx)*(-fvz(1, i, -1 + j, k, zold) + fvz(1, i, j, k, zold)) - fbz(i, j, k, bz)*(-fvz(1, i, j, -1 + k, zold) + fvz(1, i, j, k, zold)) +
		fbz(i, j, 1 + k, bz)*(-fvz(1, i, j, k, zold) + fvz(1, i, j, 1 + k, zold)) +
		fbx(i, 1 + j, k, bx)*(-fvz(1, i, j, k, zold) + fvz(1, i, 1 + j, k, zold)) +
		fby(1 + i, j, k, by)*(-fvz(1, i, j, k, zold) + fvz(1, 1 + i, j, k, zold)) -
		fky(i, j, k, ky)*(-fzz(1, -1 + i, j, k, zold) + fzz(1, i, j, k, zold)) - fkx(i, j, k, kx)*(-fzz(1, i, -1 + j, k, zold) + fzz(1, i, j, k, zold)) -
		fkz(i, j, k, kz)*(-fzz(1, i, j, -1 + k, zold) + fzz(1, i, j, k, zold)) +
		fkz(i, j, 1 + k, kz)*(-fzz(1, i, j, k, zold) + fzz(1, i, j, 1 + k, zold)) +
		fkx(i, 1 + j, k, kx)*(-fzz(1, i, j, k, zold) + fzz(1, i, 1 + j, k, zold)) +
		fky(1 + i, j, k, ky)*(-fzz(1, i, j, k, zold) + fzz(1, 1 + i, j, k, zold))) / fm(i, j, k, m);
}











__global__ void SolveKernel(int dimBlockX,int dimBlockY,int dimBlockZ,float*xoldd,float*yoldd,float*zoldd,float*xnewd,float*ynewd,float*znewd,float*md,float*kxd,float*kyd,float*kzd,float*bxd,float*byd,float*bzd,float*Fxd,float*Fyd,float*Fzd)
{
//	int tx=threadIdx.x;
//	int ty=threadIdx.y;
	int tx=blockIdx.x*dimBlockX+threadIdx.x;
	int ty=blockIdx.y*dimBlockY+threadIdx.y;
	int tz=blockIdx.z*dimBlockZ+threadIdx.z;
	
	int i=ty+1;
	int j=tx+1;
	int k=tz+1;

	xnewd[ (Ny + 2)*(4 + 2*Nx) + (k - 1)*(Ny + 2)*(4 + 2*Nx) + 4 +
		2 * Nx + (i - 1)*(4 + 2 * Nx) + 2 + (2 * j - 1) - 1] = fxx(1, i, j, k, xoldd) + fvx(1, i, j, k, xoldd)*dt;
	xnewd[ (Ny + 2)*(4 + 2 * Nx) + (k - 1)*(Ny + 2)*(4 + 2 * Nx) + 4 +
		2 * Nx + (i - 1)*(4 + 2 * Nx) + 2 + (2 * j) - 1] = fvx(1, i, j, k, xoldd) + ax(i, j, k, Fxd, xoldd, kxd, kyd, kzd, bxd, byd, bzd, md)*dt;

	ynewd[ (Nx + 2)*(4 + 2*Ny) + (k - 1)*(Nx + 2)*(4 + 2*Ny) + 4 +
		2*Ny + (j - 1)*(4 + 2*Ny) + 2 + (2*i - 1)-1] = fyy(1, i, j, k, yoldd) + fvy(1, i, j, k, yoldd)*dt;
	ynewd[ (Nx + 2)*(4 + 2*Ny) + (k - 1)*(Nx + 2)*(4 + 2*Ny) + 4 +
		2*Ny + (j - 1)*(4 + 2*Ny) + 2 + (2*i)-1] = fvy(1, i, j, k, yoldd) + ay(i, j, k, Fyd, yoldd, kxd, kyd, kzd, bxd, byd, bzd, md)*dt;

	znewd[ (Nx + 2)*(4 + 2*Nz) + (i - 1)*(Nx + 2)*(4 + 2*Nz) + 4 +
		2*Nz + (j - 1)*(4 + 2*Nz) + 2 + (2*k - 1)-1] = fzz(1, i, j, k, zoldd) + fvz(1, i, j, k, zoldd)*dt;
	znewd[ (Nx + 2)*(4 + 2*Nz) + (i - 1)*(Nx + 2)*(4 + 2*Nz) + 4 +
		2*Nz + (j - 1)*(4 + 2*Nz) + 2 + (2*k)-1] = fvz(1, i, j, k, zoldd) + az(i, j, k, Fzd, zoldd, kxd, kyd, kzd, bxd, byd, bzd, md)*dt;
}









void Solve(float*xold,float*yold,float*zold,float*xnew,float*ynew,float*znew,float*m,float*kx,float*ky,float*kz,float*bx,float*by,float*bz,float*Fx,float*Fy,float*Fz)
{
	float *xoldd,*yoldd,*zoldd,*xnewd,*ynewd,*znewd,*md,*kxd,*kyd,*kzd,*bxd,*byd,*bzd,*Fxd,*Fyd,*Fzd;
	
	int sizexoldd=xlength*sizeof(float);
	hipMalloc((void**)&xoldd,sizexoldd);
	hipMemcpy(xoldd,xold,sizexoldd,hipMemcpyHostToDevice);
	int sizeyoldd=ylength*sizeof(float);
	hipMalloc((void**)&yoldd,sizeyoldd);
	hipMemcpy(yoldd,yold,sizeyoldd,hipMemcpyHostToDevice);
	int sizezoldd=zlength*sizeof(float);
	hipMalloc((void**)&zoldd,sizezoldd);
	hipMemcpy(zoldd,zold,sizezoldd,hipMemcpyHostToDevice);
	int sizexnewd=xlength*sizeof(float);
	hipMalloc((void**)&xnewd,sizexnewd);
	hipMemcpy(xnewd,xnew,sizexnewd,hipMemcpyHostToDevice);
	int sizeynewd=ylength*sizeof(float);
	hipMalloc((void**)&ynewd,sizeynewd);
	hipMemcpy(ynewd,ynew,sizeynewd,hipMemcpyHostToDevice);
	int sizeznewd=zlength*sizeof(float);
	hipMalloc((void**)&znewd,sizeznewd);
	hipMemcpy(znewd,znew,sizeznewd,hipMemcpyHostToDevice);
	int sizemd=masslength*sizeof(float);
	hipMalloc((void**)&md,sizemd);
	hipMemcpy(md,m,sizemd,hipMemcpyHostToDevice);
	int sizekxd=kxlength*sizeof(float);
	hipMalloc((void**)&kxd,sizekxd);
	hipMemcpy(kxd,kx,sizekxd,hipMemcpyHostToDevice);
	int sizekyd=kylength*sizeof(float);
	hipMalloc((void**)&kyd,sizekyd);
	hipMemcpy(kyd,ky,sizekyd,hipMemcpyHostToDevice);
	int sizekzd=kzlength*sizeof(float);
	hipMalloc((void**)&kzd,sizekzd);
	hipMemcpy(kzd,kz,sizekzd,hipMemcpyHostToDevice);
	int sizebxd=bxlength*sizeof(float);
	hipMalloc((void**)&bxd,sizebxd);
	hipMemcpy(bxd,bx,sizebxd,hipMemcpyHostToDevice);
	int sizebyd=bylength*sizeof(float);
	hipMalloc((void**)&byd,sizebyd);
	hipMemcpy(byd,by,sizebyd,hipMemcpyHostToDevice);
	int sizebzd=bzlength*sizeof(float);
	hipMalloc((void**)&bzd,sizebzd);
	hipMemcpy(bzd,bz,sizebzd,hipMemcpyHostToDevice);
	int sizeFxd=masslength*sizeof(float);
	hipMalloc((void**)&Fxd,sizeFxd);
	hipMemcpy(Fxd,Fx,sizeFxd,hipMemcpyHostToDevice);
	int sizeFyd=masslength*sizeof(float);
	hipMalloc((void**)&Fyd,sizeFyd);
	hipMemcpy(Fyd,Fy,sizeFyd,hipMemcpyHostToDevice);
	int sizeFzd=masslength*sizeof(float);
	hipMalloc((void**)&Fzd,sizeFzd);
	hipMemcpy(Fzd,Fz,sizeFzd,hipMemcpyHostToDevice);

	//Malloc result
	//hipMalloc((void**)&Pd,size);
	//Dimensions of the run
	//int SubMtxWidth=SubWidth;
	int NBlockX=4;
	int NBlockY=3;
	int NBlockZ=5;
	int dimBlockX=Nx/NBlockX;
	int dimBlockY=Ny/NBlockY;
	int dimBlockZ=Nz/NBlockZ;
	dim3 dimBlock(dimBlockX,dimBlockY,dimBlockZ);
	dim3 dimGrid(NBlockX,NBlockY,NBlockZ);
	//Running Kernel
	SolveKernel<<<dimGrid,dimBlock>>>(dimBlockX,dimBlockY,dimBlockZ,xoldd,yoldd,zoldd,xnewd,ynewd,znewd,md,kxd,kyd,kzd,bxd,byd,bzd,Fxd,Fyd,Fzd);
	hipDeviceSynchronize();
	//Copy data back
	hipMemcpy(xnew,xnewd,sizexnewd,hipMemcpyDeviceToHost);
	hipMemcpy(ynew,ynewd,sizeynewd,hipMemcpyDeviceToHost);
	hipMemcpy(znew,znewd,sizeznewd,hipMemcpyDeviceToHost);
	checkCUDAError("memcpy");
	//Free memory
	//hipFree(Md);
	//hipFree(Nd);
	//hipFree(Pd);
	//NEWSHIT
	hipFree(xoldd);
	hipFree(yoldd);
	hipFree(zoldd);
	hipFree(xnewd);
	hipFree(ynewd);
	hipFree(znewd);
	hipFree(md);
	hipFree(kxd);
	hipFree(kyd);
	hipFree(kzd);
	hipFree(bxd);
	hipFree(byd);
	hipFree(bzd);
	hipFree(Fxd);
	hipFree(Fyd);
	hipFree(Fzd);
}
















int main(int argc,char* argv[])
{
float *xold,*yold,*zold,*xnew,*ynew,*znew,*m,*kx,*ky,*kz,*bx,*by,*bz,*Fx,*Fy,*Fz;

//----------------------------------INITIALIZATION START----------------------------------
// Solution vectors 
xold=(float *)malloc(xlength*sizeof(float));
yold=(float *)malloc(ylength*sizeof(float));
zold=(float *)malloc(zlength*sizeof(float));
xnew=(float *)malloc(xlength*sizeof(float));
ynew=(float *)malloc(ylength*sizeof(float));
znew=(float *)malloc(zlength*sizeof(float));

// Mass vector
m=(float *)malloc(masslength*sizeof(float));

// Stiffness vectors
kx=(float *)malloc(kxlength*sizeof(float));
ky=(float *)malloc(kylength*sizeof(float));
kz=(float *)malloc(kzlength*sizeof(float));

// Damping vectors
bx=(float *)malloc(bxlength*sizeof(float));
by=(float *)malloc(bylength*sizeof(float));
bz=(float *)malloc(bzlength*sizeof(float));

// Force vectors
Fx=(float *)malloc(masslength*sizeof(float));
Fy=(float *)malloc(masslength*sizeof(float));
Fz=(float *)malloc(masslength*sizeof(float));

// Initial conditions
for (int i = 0; i < xlength ; i++)
{
	xold[i]=0.0f;
	xnew[i]=0.0f;
}

for (int i = 0; i < ylength ; i++)
{
	yold[i]=0.0f;
	ynew[i]=0.0f;
}

for (int i = 0; i < zlength ; i++)
{
	zold[i]=0.0f;
	znew[i]=0.0f;
}

// Mass [kg] and forces
for (int i = 0; i < masslength ; i++)
{
	m[i]=1.0f;
	Fx[i]=0.0f;
	Fy[i]=0.0f;
	Fz[i]=0.0f;
}

// Stiffness [N/m] and damping [N sec/m] in x-direction
for (int i = 0; i < kxlength ; i++)
{
	kx[i]=0.2f;
	bx[i]=0.05f;
}

// Stiffness [N/m] and damping [N sec/m] in y-direction
for (int i = 0; i < kylength ; i++)
{
	ky[i]=0.2f;
	by[i]=0.05f;
}

// Stiffness [N/m] and damping [N sec/m] in z-direction
for (int i = 0; i < kzlength ; i++)
{
	kz[i]=0.2f;
	bz[i]=0.05f;
}
//----------------------------------INITIALIZATION END--------------------------------------



//-------------------------------BOUNDARY CONDITIONS START----------------------------------
// No connections with Top wall B.C.'s
for (int i = 1; i <= Nx; i++)
	{
		for (int k = 1; k <= Nz; k++)
		{
			ky[i + Nx*Ny + (-1 + k)*Nx*(1 + Ny) - 1] = 0.0f;
			by[i + Nx*Ny + (-1 + k)*Nx*(1 + Ny) - 1] = 0.0f;
		}
	}
//--------------------------------BOUNDARY CONDITIONS END-----------------------------------



//--------------------------------------SOLVER START-----------------------------------------
clock_t t;
t=clock();

for (int n = 1; n <= Nstep-1; n++)
	{
		// Excitation
		Fx[(2 - 1)*Ny*Nx + (6 - 1)*Nx + 8 - 1] = sin(3 * n*dtcpu); // omega = 3 [rad/sec]
		Fy[(2 - 1)*Ny*Nx + (6 - 1)*Nx + 8 - 1] = sin(3 * n*dtcpu);
		Fz[(2 - 1)*Ny*Nx + (6 - 1)*Nx + 8 - 1] = sin(3 * n*dtcpu);

		Fx[(2 - 1)*Ny*Nx + (7 - 1)*Nx + 8 - 1] = sin(3 * n*dtcpu);
		Fy[(2 - 1)*Ny*Nx + (7 - 1)*Nx + 8 - 1] = sin(3 * n*dtcpu);
		Fz[(2 - 1)*Ny*Nx + (7 - 1)*Nx + 8 - 1] = sin(3 * n*dtcpu);

		Fx[(2 - 1)*Ny*Nx + (5 - 1)*Nx + 8 - 1] = sin(3 * n*dtcpu);
		Fy[(2 - 1)*Ny*Nx + (5 - 1)*Nx + 8 - 1] = sin(3 * n*dtcpu);
		Fz[(2 - 1)*Ny*Nx + (5 - 1)*Nx + 8 - 1] = sin(3 * n*dtcpu);

		Solve(xold,yold,zold,xnew,ynew,znew,m,kx,ky,kz,bx,by,bz,Fx,Fy,Fz);
		hipDeviceSynchronize();
		// OLD=NEW
		for (int ix = 0; ix < xlength; ix++)
		{
			xold[ix] = xnew[ix];
		}
		for (int iy = 0; iy < ylength; iy++)
		{
			yold[iy] = ynew[iy];
		}
		for (int iz = 0; iz < zlength; iz++)
		{
			zold[iz] = znew[iz];
		}
	}
ofstream fout("test.txt");
if (fout.is_open())
	{
		//file opened successfully so we are here
		cout << "File Opened successfully!!!. Writing data from array to file" << endl;

		for (int j = 0; j < zlength; j++)
			{
				fout << znew[j] << ' '; //writing ith character of array in the file
			}
			fout << '\n';
		cout << "Array data successfully saved into the file test.txt" << endl;
	}
	else //file could not be opened
	{
		cout << "File could not be opened." << endl;
	}

t=clock()-t;
printf("%f seconds\n",((float)t)/CLOCKS_PER_SEC);
printf("%f,%f,%f\n",xold[60],yold[60],zold[60]);

free(xold);
free(yold);
free(zold);
free(xnew);
free(ynew);
free(znew);
free(m);
free(kx);
free(ky);
free(kz);
free(bx);
free(by);
free(bz);
free(Fx);
free(Fy);
free(Fz);

return 0;
}


void checkCUDAError(const char *msg)
{
hipError_t err = hipGetLastError();
if(hipSuccess!= err)
{
fprintf(stderr,"Cuda error: %s: %s.\n",msg,hipGetErrorString(err));
exit(EXIT_FAILURE);
}
}